#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "models.h"

#include <stdio.h>
#include "iostream"
#include "vector"
#include "ctime"

__global__ void filterKernel(LineItem *item, bool *result)
{
	int threadIndex = threadIdx.x + blockDim.x * blockIdx.x;

	result[threadIndex] = item[threadIndex].order_key == 1;
}

inline double GetElapsedTime(clock_t& since) {
	return (std::clock() - since) / (double)CLOCKS_PER_SEC * 1000;
}

std::vector<LineItem>& filter(std::vector<LineItem>& items)
{
	std::clock_t start = std::clock();

	std::vector<LineItem>& returnValue = *new std::vector<LineItem>();

	int count = items.size();

	LineItem *deviceItems;
	bool *results = false;
	bool *host_results = false;
	host_results = (bool*)malloc(count * sizeof(bool));

	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		return returnValue;
	}

	// Reserve room for input in GPU memory
	cudaStatus = hipMalloc((void**)&deviceItems, count * sizeof(LineItem));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		return returnValue;
	}

	// Copy input to GPU
	cudaStatus = hipMemcpy(deviceItems, &items[0], count * sizeof(LineItem), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		return returnValue;
	}

	// Reserve room for results in GPU memory
	cudaStatus = hipMalloc((void**)&results, count * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		return returnValue;
	}

	double duration = GetElapsedTime(start);
	std::cout << "GPU allocation and copying took " << duration << "ms\n";

	start = std::clock();

	const int threadsPerBlock = 1024;
	int blocks = (int)ceil((float)count / threadsPerBlock);

	filterKernel<TItem> <<<blocks, threadsPerBlock>>>(deviceItems, results);
	
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "filterKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return returnValue;
	}
	
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		return returnValue;
	}

	duration = GetElapsedTime(start);
	std::cout << "GPU filtering took " << duration << "ms\n";
	start = std::clock();

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(host_results, results, count * sizeof(bool), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		return returnValue;
	}

	for (int i = 0; i < count; i++)	{
		if (host_results[i]) {
			returnValue.push_back(items[i]);
		}
	}

	duration = GetElapsedTime(start);
	std::cout << "GPU reconstructing results (on CPU) took " << duration << "ms\n";

	// Cleanup
	free(host_results);
	hipFree(results);
	hipFree(deviceItems);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipFree failed\n");
		return returnValue;
	}

	return returnValue;
}

