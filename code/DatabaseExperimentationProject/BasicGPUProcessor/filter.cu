#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "stdafx.h"
#include "models.h"
#include <stdio.h>
#include <iostream>
#include <ctime>

template<typename TItem>
__global__ void filterKernel(TItem *item, bool *result, size_t totalCount) {
	//This should never be called, but exceptions are not supported; only specialized implementations allowed
}

template<> __global__ void filterKernel<LineItem>(LineItem *item, bool *result, size_t totalCount) {
	size_t threadIndex = threadIdx.x + blockDim.x * blockIdx.x;
	
	if (threadIndex < totalCount) {
		result[threadIndex] = item[threadIndex].order_key == 1;
	}
}

template<> __global__ void filterKernel<Order>(Order *item, bool *result, size_t totalCount) {
	size_t threadIndex = threadIdx.x + blockDim.x * blockIdx.x;

	if (threadIndex < totalCount) {
		result[threadIndex] = item[threadIndex].order_status == 'O';
	}
}

inline double GetElapsedTime(clock_t& since) {
	return (std::clock() - since) / (double)CLOCKS_PER_SEC * 1000;
}

void handleCudaError(hipError_t status) {
	if (status != hipSuccess) {
		fprintf(stderr, "CUDA error: %s", hipGetErrorString(status));
		hipDeviceReset();
		throw hipGetErrorString(status);
	}
}

template<typename TItem>
std::vector<TItem>& filter_standard(std::vector<TItem>& items) {
	std::clock_t start = std::clock();
	std::vector<TItem>& returnValue = *new std::vector<TItem>();
	size_t count = items.size();

	TItem *deviceItems;
	bool *deviceResults = false;
	bool *hostResults = false;
	hostResults = (bool*)malloc(count * sizeof(bool));

	// Choose which GPU to run on, change this on a multi-GPU system.
	handleCudaError(hipSetDevice(0));
	// Reserve room for input in GPU memory
	handleCudaError(hipMalloc((void**)&deviceItems, count * sizeof(TItem)));
	// Copy input to GPU
	handleCudaError(hipMemcpy(deviceItems, &items[0], count * sizeof(TItem), hipMemcpyHostToDevice));
	// Reserve room for results in GPU memory
	handleCudaError(hipMalloc((void**)&deviceResults, count * sizeof(bool)));

	std::cout << "GPU allocation and copying took " << GetElapsedTime(start) << "ms\n";
	start = std::clock();

	const int threadsPerBlock = 1024;
	int blocks = (int)ceil((float)count / threadsPerBlock);
	filterKernel<TItem> <<<blocks, threadsPerBlock>>>(deviceItems, deviceResults, count);
	
	// hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch.
	handleCudaError(hipDeviceSynchronize());

	std::cout << "GPU filtering took " << GetElapsedTime(start) << "ms\n";
	start = std::clock();

	// Copy output vector from GPU buffer to host memory.
	handleCudaError(hipMemcpy(hostResults, deviceResults, count * sizeof(bool), hipMemcpyDeviceToHost));

	for (int i = 0; i < count; i++)	{
		if (hostResults[i]) {
			returnValue.push_back(items[i]);
		}
	}

	std::cout << "GPU reconstructing results (on CPU) took " << GetElapsedTime(start) << "ms\n";

	// Cleanup
	free(hostResults);
	hipDeviceReset();

	return returnValue;
}

template std::vector<LineItem>& filter_standard<LineItem>(std::vector<LineItem>& items);
template std::vector<Order>& filter_standard<Order>(std::vector<Order>& items);

template<typename TItem>
std::vector<TItem>& filter_um(std::vector<TItem>& items) {
	std::clock_t start = std::clock();
	std::vector<TItem>& returnValue = *new std::vector<TItem>();
	size_t count = items.size();

	// Choose which GPU to run on, change this on a multi-GPU system.
	handleCudaError(hipSetDevice(0));

	TItem *managedItems;
	// Reserve room for input items in unified memory
	handleCudaError(hipMallocManaged(&managedItems, count * sizeof(TItem)));

	memcpy(managedItems, &items[0], count * sizeof(TItem));

	bool *managedResults = false;

	// Reserve room for results in unified memory
	handleCudaError(hipMallocManaged(&managedResults, count * sizeof(bool)));

	std::cout << "GPU managed allocation and copying took " << GetElapsedTime(start) << "ms\n";
	start = std::clock();

	const int threadsPerBlock = 1024;
	int blocks = (int)ceil((float)count / threadsPerBlock);
	filterKernel<TItem> << <blocks, threadsPerBlock >> >(managedItems, managedResults, count);

	// hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch.
	handleCudaError(hipDeviceSynchronize());

	std::cout << "GPU filtering took " << GetElapsedTime(start) << "ms\n";
	start = std::clock();

	for (int i = 0; i < count; i++)	{
		if (managedResults[i]) {
			returnValue.push_back(items[i]);
		}
	}

	std::cout << "GPU reconstructing results (on CPU) took " << GetElapsedTime(start) << "ms\n";

	// Cleanup
	hipFree(managedItems);
	hipFree(managedResults);
	hipDeviceReset();

	return returnValue;
}

template std::vector<LineItem>& filter_um<LineItem>(std::vector<LineItem>& items);
template std::vector<Order>& filter_um<Order>(std::vector<Order>& items);
