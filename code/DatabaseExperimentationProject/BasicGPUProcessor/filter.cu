#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "stdafx.h"
#include <stdio.h>
#include <iostream>
#include <ctime>

template<typename TItem>
__global__ void filterKernel(TItem *item, bool *result, size_t totalCount) {
	//This should never be called, but exceptions are not supported; only specialized implementations allowed
}

template<> __global__ void filterKernel<LineItem>(LineItem *item, bool *result, size_t totalCount) {
	size_t threadIndex = threadIdx.x + blockDim.x * blockIdx.x;
	
	if (threadIndex < totalCount) {
		result[threadIndex] = item[threadIndex].order_key == 1;
	}
}

template<> __global__ void filterKernel<Order>(Order *item, bool *result, size_t totalCount) {
	size_t threadIndex = threadIdx.x + blockDim.x * blockIdx.x;

	if (threadIndex < totalCount) {
		result[threadIndex] = item[threadIndex].order_status == 'O';
	}
}

inline double GetElapsedTime(clock_t& since) {
	return (std::clock() - since) / (double)CLOCKS_PER_SEC * 1000;
}

void handleCudaError(hipError_t status) {
	if (status != hipSuccess) {
		fprintf(stderr, "CUDA error: %s", hipGetErrorString(status));
		hipDeviceReset();
		throw hipGetErrorString(status);
	}
}

template<typename TItem>
std::vector<TItem>& filter_standard(std::vector<TItem>& items) {
	std::clock_t start = std::clock();
	std::vector<TItem>& returnValue = *new std::vector<TItem>();
	size_t count = items.size();

	TItem *deviceItems;
	bool *deviceResults = false;
	bool *hostResults = false;
	hostResults = (bool*)malloc(count * sizeof(bool));

	// Choose which GPU to run on, change this on a multi-GPU system.
	handleCudaError(hipSetDevice(0));
	// Reserve room for input in GPU memory
	handleCudaError(hipMalloc((void**)&deviceItems, count * sizeof(TItem)));
	// Copy input to GPU
	handleCudaError(hipMemcpy(deviceItems, &items[0], count * sizeof(TItem), hipMemcpyHostToDevice));
	// Reserve room for results in GPU memory
	handleCudaError(hipMalloc((void**)&deviceResults, count * sizeof(bool)));

	std::cout << "GPU allocation and copying took " << GetElapsedTime(start) << "ms\n";
	start = std::clock();

	const int threadsPerBlock = 1024;
	int blocks = (int)ceil((float)count / threadsPerBlock);
	filterKernel<TItem> <<<blocks, threadsPerBlock>>>(deviceItems, deviceResults, count);
	
	// hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch.
	handleCudaError(hipDeviceSynchronize());

	std::cout << "GPU filtering took " << GetElapsedTime(start) << "ms\n";
	start = std::clock();

	// Copy output vector from GPU buffer to host memory.
	handleCudaError(hipMemcpy(hostResults, deviceResults, count * sizeof(bool), hipMemcpyDeviceToHost));

	for (unsigned int i = 0; i < count; i++)	{
		if (hostResults[i]) {
			returnValue.push_back(items[i]);
		}
	}

	std::cout << "GPU reconstructing results (on CPU) took " << GetElapsedTime(start) << "ms\n";

	// Cleanup
	free(hostResults);
	hipDeviceReset();

	return returnValue;
}

template std::vector<LineItem>& filter_standard<LineItem>(std::vector<LineItem>& items);
template std::vector<Order>& filter_standard<Order>(std::vector<Order>& items);

template<typename TItem>
std::vector<TItem>& filter_um(std::vector<TItem>& items) {
	std::clock_t start = std::clock();
	std::vector<TItem>& returnValue = *new std::vector<TItem>();
	size_t count = items.size();

	// Choose which GPU to run on, change this on a multi-GPU system.
	handleCudaError(hipSetDevice(0));

	TItem *managedItems;
	// Reserve room for input items in unified memory
	handleCudaError(hipMallocManaged(&managedItems, count * sizeof(TItem)));

	memcpy(managedItems, &items[0], count * sizeof(TItem));

	bool *managedResults = false;

	// Reserve room for results in unified memory
	handleCudaError(hipMallocManaged(&managedResults, count * sizeof(bool)));

	std::cout << "GPU managed allocation and copying took " << GetElapsedTime(start) << "ms\n";
	start = std::clock();

	const int threadsPerBlock = 1024;
	int blocks = (int)ceil((float)count / threadsPerBlock);
	filterKernel<TItem> << <blocks, threadsPerBlock >> >(managedItems, managedResults, count);

	// hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch.
	handleCudaError(hipDeviceSynchronize());

	std::cout << "GPU filtering took " << GetElapsedTime(start) << "ms\n";
	start = std::clock();

	for (unsigned int i = 0; i < count; i++) {
		if (managedResults[i]) {
			returnValue.push_back(items[i]);
		}
	}

	std::cout << "GPU reconstructing results (on CPU) took " << GetElapsedTime(start) << "ms\n";

	// Cleanup
	hipFree(managedItems);
	hipFree(managedResults);
	hipDeviceReset();

	return returnValue;
}

template std::vector<LineItem>& filter_um<LineItem>(std::vector<LineItem>& items);
template std::vector<Order>& filter_um<Order>(std::vector<Order>& items);

template<typename TItem>
std::vector<TItem>& filter_async(std::vector<TItem>& items) {
	std::clock_t start = std::clock();
	std::vector<TItem>& returnValue = *new std::vector<TItem>();
	size_t count = items.size();

	TItem *deviceItems;
	TItem *pinnedItems;
	bool *deviceResults = false;
	bool *hostResults = false;
	hostResults = (bool*)malloc(count * sizeof(bool));
	// Choose which GPU to run on, change this on a multi-GPU system.
	handleCudaError(hipSetDevice(0));

	// Prepare streams
	const int streamCount = 4;
	hipStream_t *streams;
	streams = (hipStream_t*)malloc(streamCount * sizeof(hipStream_t));
	for (int i = 0; i < streamCount; i++) {
		handleCudaError(hipStreamCreate(&streams[i]));
	}

	// Reserve pinned host memory for data
	handleCudaError(hipHostMalloc((void**)&pinnedItems, count * sizeof(TItem)));
	// Copy input to pinned memory
	memcpy(pinnedItems, &items[0], count * sizeof(TItem));

	// Reserve room for input in GPU memory
	handleCudaError(hipMalloc((void**)&deviceItems, count * sizeof(TItem)));
	// Reserve room for results in GPU memory
	handleCudaError(hipMalloc((void**)&deviceResults, count * sizeof(bool)));

	size_t perStreamCount = count / streamCount;
	const int threadsPerBlock = 1024;
	int blocks = (int)ceil((float)count / threadsPerBlock);

	/* Depending on the GPU's capabilities this way of calling or calling all three CUDA functions in a single loop might be faster.
	   For details, see: http://devblogs.nvidia.com/parallelforall/how-overlap-data-transfers-cuda-cc/ */
	// Copy input to GPU
	for (int i = 0; i < streamCount; i++) {
		size_t transferCount = perStreamCount;
		if (i == streamCount - 1) transferCount = count - i * transferCount;

		handleCudaError(hipMemcpyAsync(&deviceItems[i*perStreamCount], &pinnedItems[i*perStreamCount], transferCount * sizeof(TItem), hipMemcpyHostToDevice, streams[i]));
	}
	// Execute kernels
	for (int i = 0; i < streamCount; i++) {
		size_t transferCount = perStreamCount;
		if (i == streamCount - 1) transferCount = count - i * transferCount;

		filterKernel<TItem> << <blocks, threadsPerBlock, 0, streams[i] >> >(&deviceItems[i*perStreamCount], &deviceResults[i*perStreamCount], transferCount);
	}
	// Copy output vector from GPU buffer to host memory.
	for (int i = 0; i < streamCount; i++) {
		size_t transferCount = perStreamCount;
		if (i == streamCount - 1) transferCount = count - i * transferCount;

		handleCudaError(hipMemcpyAsync(&hostResults[i*perStreamCount], &deviceResults[i*perStreamCount], transferCount * sizeof(bool), hipMemcpyDeviceToHost, streams[i]));
	}

	// hipDeviceSynchronize waits for the kernel and copy operations to finish, and returns any errors encountered during the launch.
	handleCudaError(hipDeviceSynchronize());

	for (int i = 0; i < streamCount; i++) {
		handleCudaError(hipStreamDestroy(streams[i]));
	}

	std::cout << "GPU allocation, copying and filtering took " << GetElapsedTime(start) << "ms\n";
	start = std::clock();

	for (unsigned int i = 0; i < count; i++) {
		if (hostResults[i]) {
			returnValue.push_back(items[i]);
		}
	}

	std::cout << "GPU reconstructing results (on CPU) took " << GetElapsedTime(start) << "ms\n";

	// Cleanup
	free(hostResults);
	hipHostFree(pinnedItems);
	hipDeviceReset();

	return returnValue;
}

template std::vector<LineItem>& filter_async<LineItem>(std::vector<LineItem>& items);
template std::vector<Order>& filter_async<Order>(std::vector<Order>& items);
