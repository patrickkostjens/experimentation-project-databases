#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "cuda_helpers.cuh"

double GetElapsedTime(clock_t& since) {
	return (std::clock() - since) / (double)CLOCKS_PER_SEC * 1000;
}

void handleCudaError(hipError_t status) {
	if (status != hipSuccess) {
		fprintf(stderr, "CUDA error: %s", hipGetErrorString(status));
		hipDeviceReset();
		throw hipGetErrorString(status);
	}
}
